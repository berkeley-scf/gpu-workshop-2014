
#include <hip/hip_runtime.h>
#define SQRT_TWO_PI 2.506628274631000
extern "C"
__global__ void calc_loglik(double* vals, int N, double mu, double sigma) {
   // note that this assumes no third dimension to the grid
     // id of the block
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    // size of each block (within grid of blocks)
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    // id of thread in a given block
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    // assign overall id/index of the thread
    int idx = myblock * blocksize + subthread;

        if(idx < N) {
            double std = (vals[idx] - mu)/ sigma;
            double e = exp( - 0.5 * std * std);
            vals[idx] = e / ( sigma * SQRT_TWO_PI);
        }
}
