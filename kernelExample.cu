#include <stdlib.h>
#include <sys/time.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

//#define N 1000000
#define SQRT_TWO_PI 2.506628274631000
#define BLOCK_D1 1024
#define BLOCK_D2 1
#define BLOCK_D3 1

// Note: Needs compute capability >= 2.0 for calculation with doubles, so compile with:
// nvcc kernelExample.cu -arch=compute_20 -code=sm_20,compute_20 -o kernelExample
// -use_fast_math doesn't seem to have any effect on speed

// CUDA kernel:
__global__ void calc_loglik(double* vals, int N, double mu, double sigma) {
   // note that this assumes no third dimension to the grid
    // id of the block
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    // size of each block (within grid of blocks)
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    // id of thread in a given block
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    // assign overall id/index of the thread
    int idx = myblock * blocksize + subthread;

        if(idx < N) {
            double std = (vals[idx] - mu)/sigma;
            double e = exp( - 0.5 * std * std);
            vals[idx] = e / ( sigma * SQRT_TWO_PI);
        }
}

// CPU analog for speed comparison
int calc_loglik_cpu(double* vals, int N, double mu, double sigma) {
  double std, e;
  for(int idx = 0; idx < N; idx++) {
    std = (vals[idx] - mu)/sigma;
    e = exp( - 0.5 * std * std);
    vals[idx] = e / ( sigma * SQRT_TWO_PI);
  }
  return 0;
}


/* --------------------------- host code ------------------------------*/
void fill( double *p, int n ) {
  int i;
  srand48(0);
  for( i = 0; i < n; i++ )
    p[i] = 2*drand48()-1;
}

double read_timer() {
  struct timeval end;
  gettimeofday( &end, NULL );
  return end.tv_sec+1.e-6*end.tv_usec;
}

int main (int argc, char *argv[]) {
  double* cpu_vals;
  double* gpu_vals;
  int N;
  hipError_t cudaStat;
 
  printf("====================================================\n");
  for( N = 32768; N <= 134217728; N*=8 ) {
    cpu_vals = (double*) malloc( sizeof(double)*N );
    cudaStat = hipMalloc(&gpu_vals, sizeof(double)*N);
    if(cudaStat != hipSuccess) {
      printf ("device memory allocation failed");
      return EXIT_FAILURE;
    }

    // fixed block dimensions (1024x1x1 threads)
    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    
    // determine number of blocks we need for a given problem size
    int tmp = ceil(pow(N/BLOCK_D1, 0.5));
    printf("Grid dimension is %i x %i\n", tmp, tmp);
    dim3 gridSize(tmp, tmp, 1);

    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*tmp*tmp;
    if (nthreads < N){
        printf("\n============ NOT ENOUGH THREADS TO COVER N=%d ===============\n\n",N);
    } else {
        printf("Launching %d threads (N=%d)\n", nthreads, N);
    }

    double mu = 0.0;
    double sigma = 1.0;

    // simulate 'data'
    fill(cpu_vals, N);
    printf("Input values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    hipDeviceSynchronize();
    double tInit = read_timer();

    // copy input data to the GPU
    cudaStat = hipMemcpy(gpu_vals, cpu_vals, N*sizeof(double), hipMemcpyHostToDevice);
    printf("Memory Copy from Host to Device ");
    if (cudaStat){
      printf("failed.\n");
    } else {
      printf("successful.\n");
    }
    hipDeviceSynchronize();
    double tTransferToGPU = read_timer();

    // do the calculation
    calc_loglik<<<gridSize, blockSize>>>(gpu_vals, N, mu, sigma);
    
    hipDeviceSynchronize();
    double tCalc = read_timer();

    cudaStat = hipMemcpy(cpu_vals, gpu_vals, N, hipMemcpyDeviceToHost);
    printf("Memory Copy from Device to Host ");
    if (cudaStat){
      printf("failed.\n");
    } else {
      printf("successful.\n");
    }
    hipDeviceSynchronize();
    double tTransferFromGPU = read_timer();

    printf("Output values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    // do calculation on CPU for comparison (unfair as this will only use one core)
    fill(cpu_vals, N);
    double tInit2 = read_timer();
    calc_loglik_cpu(cpu_vals, N, mu, sigma);
    double tCalcCPU = read_timer();

    printf("Output values (CPU): %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    printf("Timing results for n = %d\n", N);
    printf("Transfer to GPU time: %f\n", tTransferToGPU - tInit);
    printf("Calculation time (GPU): %f\n", tCalc - tTransferToGPU);
    printf("Calculation time (CPU): %f\n", tCalcCPU - tInit2);
    printf("Transfer from GPU time: %f\n", tTransferFromGPU - tCalc);

    printf("Freeing memory...\n");
    printf("====================================================\n");
    free(cpu_vals);
    hipFree(gpu_vals);

  }
  printf("\n\nFinished.\n\n");
  return 0;
}

 
