#include <stdlib.h>
#include <sys/time.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

//#define N 1000000
#define SQRT_TWO_PI 2.506628274631000
#define BLOCK_D1 1024
#define BLOCK_D2 1
#define BLOCK_D3 1

// Note: Needs compute capability > 2.0 for calculation with doubles, so compile with:
// nvcc kernelExample-pinned.cu -arch=compute_20 -code=sm_20,compute_20 -o kernelExample-pinned
// -use_fast_math

// CUDA kernel:
__global__ void calc_loglik(double* vals, int N, double mu, double sigma) {
   // note that this assumes no third dimension to the grid
    // id of the block
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    // size of each block (within grid of blocks)
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    // id of thread in a given block
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    // assign overall id/index of the thread
    int idx = myblock * blocksize + subthread;

        if(idx < N) {
            double std = (vals[idx] - mu)/sigma;
            double e = exp( - 0.5 * std * std);
            vals[idx] = e / ( sigma * SQRT_TWO_PI);
        }
}

int calc_loglik_cpu(double* vals, int N, double mu, double sigma) {
  double std, e;
  for(int idx = 0; idx < N; idx++) {
    std = (vals[idx] - mu)/sigma;
    e = exp( - 0.5 * std * std);
    vals[idx] = e / ( sigma * SQRT_TWO_PI);
  }
  return 0;
}


/* --------------------------- host code ------------------------------*/
void fill( double *p, int n ) {
  int i;
  srand48(0);
  for( i = 0; i < n; i++ )
    p[i] = 2*drand48()-1;
}

double read_timer() {
  struct timeval end;
  gettimeofday( &end, NULL );
  return end.tv_sec+1.e-6*end.tv_usec;
}

int main (int argc, char *argv[]) {
  double* cpu_vals;
  double* gpu_vals;
  int N;
  hipError_t cudaStat;
 
  printf("====================================================\n");
  for( N = 32768; N <= 134217728; N*=8 ) {
    // allocated pinned and mapped memory on CPU
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc((void**)&cpu_vals, N*sizeof(double), hipHostMallocMapped);

    // map the CPU storage to the GPU to the CPU storage
    cudaStat = hipHostGetDevicePointer(&gpu_vals, cpu_vals, 0);

    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    
    int tmp = ceil(pow(N/BLOCK_D1, 0.5));
    printf("Grid dimension is %i x %i\n", tmp, tmp);
    dim3 gridSize(tmp, tmp, 1);

    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*tmp*tmp;
    if (nthreads < N){
        printf("\n============ NOT ENOUGH THREADS TO COVER N=%d ===============\n\n",N);
    } else {
        printf("Launching %d threads (N=%d)\n", nthreads, N);
    }

    double mu = 0.0;
    double sigma = 1.0;

    // simulate 'data'
    fill(cpu_vals, N);
    printf("Input values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    hipDeviceSynchronize();
    double tInit = read_timer();

    // do the calculation
    calc_loglik<<<gridSize, blockSize>>>(gpu_vals, N, mu, sigma);
    
    hipDeviceSynchronize();
    double tCalc = read_timer();

    printf("Output values: %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    // do calculation on CPU for comparison (unfair as this will only use one core)
    fill(cpu_vals, N);
    double tInit2 = read_timer();
    calc_loglik_cpu(cpu_vals, N, mu, sigma);
    double tCalcCPU = read_timer();

    printf("Output values (CPU): %f %f %f...\n", cpu_vals[0], cpu_vals[1], cpu_vals[2]);

    printf("Timing results for n = %d\n", N);
    printf("Calculation time (GPU): %f\n", tCalc - tInit);
    printf("Calculation time (CPU): %f\n", tCalcCPU - tInit2);

    printf("Freeing memory...\n");
    printf("====================================================\n");
    hipHostFree(cpu_vals);

  }
  printf("\n\nFinished.\n\n");
  return 0;
}

